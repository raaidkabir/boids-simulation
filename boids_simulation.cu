#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include "imgui.h"
#include "imgui_impl_glut.h"
#include "imgui_impl_opengl2.h"

const float maxSpeed = 2.0f;
const float maxForce = 0.5f;
const int numBoids = 500;

// Global variables for ImGui
float visualRange = 50.0f;
float cohesion = 0.0f;
float separation = 0.0f;
float alignment = 0.0f;

struct Boid {
    float x, y, z;
    float vx, vy, vz;
};

Boid* d_boids;
std::vector<Boid> boids;

__device__ void limitVector(float& x, float& y, float& z, float max) {
    float mag = sqrt(x * x + y * y + z * z);
    if (mag > max) {
        x = (x / mag) * max;
        y = (y / mag) * max;
        z = (z / mag) * max;
    }
}

__device__ void applyBoundary(Boid& boid, float minX, float maxX, float minY, float maxY, float minZ, float maxZ, float margin) {
    if (boid.x < minX + margin) boid.vx += maxForce;
    else if (boid.x > maxX - margin) boid.vx -= maxForce;

    if (boid.y < minY + margin) boid.vy += maxForce;
    else if (boid.y > maxY - margin) boid.vy -= maxForce;

    if (boid.z < minZ + margin) boid.vz += maxForce;
    else if (boid.z > maxZ - margin) boid.vz -= maxForce;
}

__global__ void applyRulesKernel(Boid* d_boids, int numBoids, float visualRange, float cohesion, float separation, float alignment) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numBoids) return;

    Boid boid = d_boids[i];
    float sepX = 0.0f, sepY = 0.0f, sepZ = 0.0f;
    float alignX = 0.0f, alignY = 0.0f, alignZ = 0.0f;
    float cohX = 0.0f, cohY = 0.0f, cohZ = 0.0f;
    int count = 0;

    for (int j = 0; j < numBoids; ++j) {
        if (i == j) continue;
        Boid other = d_boids[j];
        float dx = other.x - boid.x;
        float dy = other.y - boid.y;
        float dz = other.z - boid.z;
        float distance = sqrt(dx * dx + dy * dy + dz * dz);

        if (distance > 0 && distance < visualRange) {
            sepX -= dx / distance; // Separation
            sepY -= dy / distance;
            sepZ -= dz / distance;
            alignX += other.vx;    // Alignment
            alignY += other.vy;
            alignZ += other.vz;
            cohX += other.x;       // Cohesion
            cohY += other.y;
            cohZ += other.z;
            count++;
        }
    }

    if (count > 0) {
        alignX /= count;
        alignY /= count;
        alignZ /= count;
        cohX /= count;
        cohY /= count;
        cohZ /= count;

        alignX = (alignX - boid.vx) * alignment;
        alignY = (alignY - boid.vy) * alignment;
        alignZ = (alignZ - boid.vz) * alignment;
        cohX = (cohX - boid.x) * cohesion;
        cohY = (cohY - boid.y) * cohesion;
        cohZ = (cohZ - boid.z) * cohesion;

        sepX *= separation;
        sepY *= separation;
        sepZ *= separation;

        limitVector(alignX, alignY, alignZ, maxForce);
        limitVector(cohX, cohY, cohZ, maxForce);
        limitVector(sepX, sepY, sepZ, maxForce);

        boid.vx += alignX + cohX + sepX;
        boid.vy += alignY + cohY + sepY;
        boid.vz += alignZ + cohZ + sepZ;
    }

    limitVector(boid.vx, boid.vy, boid.vz, maxSpeed);
    applyBoundary(boid, 0, 800, 0, 600, 0, 600, 50); // Apply soft boundaries
    boid.x += boid.vx;
    boid.y += boid.vy;
    boid.z += boid.vz;

    __syncthreads();

    d_boids[i] = boid;
}

void initializeBoids(int numBoids) {
    boids.clear();
    for (int i = 0; i < numBoids; ++i) {
        Boid boid = {
            static_cast<float>(rand() % 800), // x position
            static_cast<float>(rand() % 600), // y position
            static_cast<float>(rand() % 600), // z position
            (static_cast<float>(rand()) / RAND_MAX) * 2 - 1, // x velocity
            (static_cast<float>(rand()) / RAND_MAX) * 2 - 1, // y velocity
            (static_cast<float>(rand()) / RAND_MAX) * 2 - 1  // z velocity
        };
        boids.push_back(boid);
    }
    hipMalloc(&d_boids, numBoids * sizeof(Boid));
    hipMemcpy(d_boids, boids.data(), numBoids * sizeof(Boid), hipMemcpyHostToDevice);
}

void applyRules(int numBoids, float cohesion, float separation, float alignment) {
    int blockSize = 1024;
    int numBlocks = (numBoids + blockSize - 1) / blockSize;
    applyRulesKernel<<<numBlocks, blockSize>>>(d_boids, numBoids, visualRange, cohesion, separation, alignment);
    hipMemcpy(boids.data(), d_boids, numBoids * sizeof(Boid), hipMemcpyDeviceToHost);
}

void drawPyramid(Boid boid) {
    float size = 5.0f; // size of the pyramid

    // Calculate the normalized velocity vector
    float mag = sqrt(boid.vx * boid.vx + boid.vy * boid.vy + boid.vz * boid.vz);
    float nx = boid.vx / mag;
    float ny = boid.vy / mag;
    float nz = boid.vz / mag;

    // Base vertices of the pyramid
    float base1[3] = {boid.x - size, boid.y - size, boid.z - size};
    float base2[3] = {boid.x + size, boid.y - size, boid.z - size};
    float base3[3] = {boid.x, boid.y + size, boid.z - size};

    // Tip of the pyramid pointing in the direction of the velocity vector
    float tip[3] = {boid.x + nx * size * 2, boid.y + ny * size * 2, boid.z + nz * size * 2};

    glBegin(GL_TRIANGLES);

    // Base of the pyramid
    glColor3f(0.0f, 0.0f, 1.0f); 
    glVertex3fv(base1);
    glVertex3fv(base2);
    glVertex3fv(base3);

    // Sides of the pyramid
    glColor3f(0.0f, 0.0f, 1.0f); 
    glVertex3fv(base1);
    glVertex3fv(base2);
    glVertex3fv(tip);

    glColor3f(0.0f, 1.0f, 0.0f); 
    glVertex3fv(base2);
    glVertex3fv(base3);
    glVertex3fv(tip);

    glColor3f(1.0f, 0.0f, 0.0f);
    glVertex3fv(base3);
    glVertex3fv(base1);
    glVertex3fv(tip);

    glEnd();
}

void display() {
    // Start the ImGui frame
    ImGui_ImplOpenGL2_NewFrame();
    ImGui_ImplGLUT_NewFrame();
    ImGui::NewFrame();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glLoadIdentity();
    gluLookAt(400, 300, 600, 400, 300, 0, 0, 1, 0); // Adjust camera position

    for (const auto& boid : boids) {
        drawPyramid(boid);
    }

    // Create ImGui window
    ImGui::Begin("Boids Parameters");
    ImGui::SliderFloat("Visual Range", &visualRange, 50, 500);
    ImGui::SliderFloat("Cohesion", &cohesion, -0.2f, 0.1f);
    ImGui::SliderFloat("Separation", &separation, -0.2f, 0.1f);
    ImGui::SliderFloat("Alignment", &alignment, -0.2f, 0.1f);
    ImGui::End();

    // Rendering ImGui
    ImGui::EndFrame();
    ImGui::Render();
    ImGui_ImplOpenGL2_RenderDrawData(ImGui::GetDrawData());

    glutSwapBuffers();
}

void update(int value) {
    applyRules(numBoids, cohesion, separation, alignment);
    glutPostRedisplay();
    glutTimerFunc(16, update, 0);
}

void init() {
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glEnable(GL_DEPTH_TEST);
    glMatrixMode(GL_PROJECTION);
    gluPerspective(45.0, 4.0 / 3.0, 1.0, 1000.0);
    glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(800, 600);
    glutCreateWindow("3D Boids Flocking Simulation");

    // Initialize ImGui
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    ImGui::StyleColorsDark();
    ImGui_ImplGLUT_Init();
    ImGui_ImplGLUT_InstallFuncs();
    ImGui_ImplOpenGL2_Init();

    init();
    initializeBoids(numBoids);

    glutDisplayFunc(display);
    glutTimerFunc(16, update, 0);
    glutMainLoop();

    hipFree(d_boids);

    // Cleanup ImGui
    ImGui_ImplOpenGL2_Shutdown();
    ImGui_ImplGLUT_Shutdown();
    ImGui::DestroyContext();

    return 0;
}
